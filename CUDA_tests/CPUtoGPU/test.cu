
#include <cstring>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mpi.h"
#include "cudaCheck.h"




// number of elements of the host buffer. All of them are sent to the device. 
#define N 1
// number of elements that will be received on the dev buffer. M must be <= N.

// When N = M ~ 1e5             -> Everything works.
// When N = M < 4.2e4 (0.84 MB) -> Segmentation fault (non UCX related)
// When N = M > 5.7e7 (114 MB)  -> UCX ERROR: Segmentation fault.
// When N < 4.2e4 and M << N    -> works fine. 

// 2147483647


//int main()
int main(int argc, const char * argv[])
{

    int size = argc > 1 ? atoi(argv[1]) : N;
    
    MPI_Init(nullptr, nullptr);
    
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    
    std::cout << "hi from rank " << rank << std::endl;

    int *buf_host ;
    int *buf_dev  ;

    // the size of both buffers (not necessarily the size of the package)
    int bufferSize = size*sizeof(int);

    buf_host = (int*)malloc(bufferSize);
    
    cudaCheck( hipMalloc((void**)&buf_dev, bufferSize) );
    cudaCheck( hipMemset(buf_dev, 0, bufferSize)       );
    

    MPI_Barrier(MPI_COMM_WORLD);
    
    if (rank == 0) {
        printf("-------->     r0_0\n");
        //MPI_Send(buf_host, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD);
        MPI_Recv(buf_host, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        //MPI_Send(buf_dev, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD);
        //MPI_Recv(buf_dev, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        printf("-------->     r0_1\n");
    } else { 
        // receive into the device buffer
        printf("-------->     r1_0\n"); 
        //MPI_Recv(buf_dev, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        MPI_Ssend(buf_dev, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD);
        //MPI_Recv(buf_host, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        //MPI_Send(buf_host, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD);
        printf("-------->     r1_1\n");
    }

    if (rank == 0) {std::cout << buf_host[0] << std::endl;}
    
    //free(buf_host);
	//hipFree(buf_dev);
    
    std::cout << "rank " << rank << " done" << std::endl;
    
    MPI_Finalize();
}

