
#include <cstring>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mpi.h"





// number of elements of the host buffer. All of them are sent to the device. 
#define N 1
// number of elements that will be received on the dev buffer. M must be <= N.

// When N = M ~ 1e5             -> Everything works.
// When N = M < 4.2e4 (0.84 MB) -> Segmentation fault (non UCX related)
// When N = M > 5.7e7 (114 MB)  -> UCX ERROR: Segmentation fault.
// When N < 4.2e4 and M << N    -> works fine. 

// 2147483647

/* After printing some memory addresses, I have noticed that the segmentation
faults occur because MPI is still trying to write in the device after the buffer
is over. Since the floats weight 4 bytes both in Patatrack and in Felk, it could
be most likely because MPI doesn't start writing the buffer right at the
beginning. I am still trying to prove this.*/


//int main()
int main(int argc, const char * argv[])
{

    printf("hello\n");

    int rank;
    int size = argc > 1 ? atoi(argv[1]) : N;
    //int size = N;
    
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    
    float *buf_host ;
    float *buf_dev  ;

    // the size of both buffers (not necessarily the size of the package)
    int bufferSize = size*sizeof(float);

    buf_host = (float*)malloc(bufferSize);

	hipMalloc(&buf_dev, bufferSize);               // works with small pkgs    

    if (rank == 0) {
        printf("-------->     r0_0\n");
        //MPI_Send(buf_host, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD);
        MPI_Recv(buf_host, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        //MPI_Send(buf_dev, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD);
        //MPI_Recv(buf_dev, size, MPI_FLOAT, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        printf("-------->     r0_1\n");
    } else { 
        // receive into the device buffer
        printf("-------->     r1_0\n");
        //MPI_Recv(buf_dev, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        MPI_Send(buf_dev, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD);
        //MPI_Recv(buf_host, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        //MPI_Send(buf_host, size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD);
        printf("-------->     r1_1\n");
    }


    free(buf_host);
	hipFree(buf_dev);
    
    printf("done\n");
    MPI_Finalize();
}

