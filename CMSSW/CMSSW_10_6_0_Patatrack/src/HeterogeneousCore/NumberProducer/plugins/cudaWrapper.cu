#include "hip/hip_runtime.h"
/*

    This file defines a simple kernel that sums all the elements of a given
    memory buffer.

*/ 



#include <hip/hip_runtime.h>
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include <iostream>
#include "header.h"



//void AccumulatorGPU(double* sum_d)
__global__ 
void AccumulatorGPU(double* buf_d, double* sum_d, int len)
{
    sum_d[0] = 0;

    for (int i = 0; i < len; i++)
    {
        sum_d[0] += buf_d[i];
    }

    printf("[cudaWrapper::AccumulatorGPU]: Result: %.5f\n", sum_d[0]);
}

void cudaWrapper(double* buf_d, double* sum_d, int len)
{
    
    cudaCheck( hipDeviceSynchronize()); // wait for everyone
    LOG("[cudaWrapper::cudaWrapper]:  calling the kernel", 1);
    AccumulatorGPU<<<1, 1>>>(buf_d, sum_d, len);
    LOG("[cudaWrapper::cudaWrapper]:  kernel called", 1);
    cudaCheck( hipDeviceSynchronize() ); // wait for everyone
    LOG("[cudaWrapper::cudaWrapper]:  kernel finished", 1);
}
